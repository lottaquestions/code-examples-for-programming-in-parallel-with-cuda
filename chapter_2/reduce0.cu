#include "hip/hip_runtime.h"
#include "cx.h"
#include "cxtimers.h"
#include <random>


// To compile: nvcc -I/home/lottaquestions/nvidia-installers/cuda-samples/Common -G -o reduce0.bin reduce0.cu

__global__ void reduce0(float *x, int m){
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    x[tid] += x[tid + m];
}

int main(int argc, char *argv[]){
    int N = (argc > 1) ? atoi(argv[1]) : 1 << 24; // 2^24

    thrust::host_vector<float> x(N);
    thrust::device_vector<float> dev_x(N);

    // Initialize x with random numbers and copy to dx
    std::default_random_engine gen(12345678);
    std::uniform_real_distribution<float> fran(0.0, 1.0);
    for (int k = 0; k < N; k++)
        x[k] = fran(gen);

    dev_x = x; // Host to device copy of N words.

    cx::timer tim;
    // Host reduce
    double host_sum = 0.0;
    for(int k = 0; k < N; k++) 
        host_sum += x[k];
    double t1 = tim.lap_ms();

    // Simple GPU reduce for N = power of 2
    tim.reset();

    for(int m = N/2; m > 0; m/=2){
        int threads = std::min(256, m);
        int blocks = std::max(m/256, 1);
        reduce0<<<blocks, threads>>>(dev_x.data().get(), m);
    }
    hipDeviceSynchronize(); // Causes host to wait for all pending GPU operations to complete before continuing.
    double t2 = tim.lap_ms();

    double gpu_sum = dev_x[0]; // Device to host copy (1 word)
    printf("sum of %d random numbers: host %.1f %.3f ms, GPU %.1f %.3f ms\n", N, host_sum, t1, gpu_sum, t2);
    return 0;
}