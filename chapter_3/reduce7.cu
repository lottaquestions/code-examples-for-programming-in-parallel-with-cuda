#include "hip/hip_runtime.h"
#include "hip/hip_cooperative_groups.h"
#include "cx.h"
#include "cxtimers.h"
#include <random>

// To compile: nvcc -I/home/lottaquestions/nvidia-installers/cuda-samples/Common -G -o reduce7.bin reduce7.cu

namespace cg = cooperative_groups;

// No shared memory is used in this implementation and hence allows maximal use of L1 cache.

__global__ void reduce7(r_Ptr<float> sums, cr_Ptr<float> data, int n){
    // This kernel assumes array sums is set to zero on entry and that
    // blockSize is a multiple of 32
    auto grid = cg::this_grid();
    auto block = cg::this_thread_block();
    auto warp = cg::tiled_partition<32>(block);

    // Accumulate thread sums in register variable v
    float v = 0.0f;
    for(int tid = grid.thread_rank(); tid < n ; tid += grid.size())
        v += data[tid];
    warp.sync();
    v += warp.shfl_down(v, 16); // | 
    v += warp.shfl_down(v, 8);  // | 
    v += warp.shfl_down(v, 4);  // | Warp level reduce here
    v += warp.shfl_down(v, 2);  // | 
    v += warp.shfl_down(v, 1);  // | 

    // Use atomicAdd to sum over warps
    if (warp.thread_rank() == 0)
        atomicAdd(&sums[block.group_index().x], v);
}

int main(int argc,char *argv[])
{
	int N       = (argc > 1) ? 1 << atoi(argv[1]) : 1 << 24; // default 2^24
	int blocks  = (argc > 2) ? atoi(argv[2]) : 256;
	int threads = (argc > 3) ? atoi(argv[3]) : 256;  // multiple of 32
	int nreps   = (argc > 4) ? atoi(argv[4]) : 1; // set this to 1 for correct answer or >> 1 for timing tests
	thrust::host_vector<float>    x(N);
	thrust::device_vector<float>  dx(N);
	thrust::device_vector<float>  dy(blocks);

	// initialise x with random numbers and copy to dx.
	std::default_random_engine gen(12345678);
	std::uniform_real_distribution<float> fran(0.0,1.0);
	for(int k = 0; k<N; k++) x[k] = fran(gen);
	dx = x;  // H2D copy (N words)
	cx::timer tim;
	double host_sum = 0.0;
	for(int k = 0; k<N; k++) host_sum += x[k]; // host reduce!
	double t1 = tim.lap_ms();

	tim.reset();
	// NB tacit assumtion that output array preset to zero. This is only needed to get correct result
	// for case nreps=1. Larger values of nreps are only used for timing purposes.	
	for(int rep=0;rep<nreps;rep++){
		reduce7<<<blocks,threads,threads*sizeof(float)>>>(dy.data().get(),dx.data().get(),N);
	}
	// use reduce7 for both steps.
	reduce7<<<1,blocks,blocks*sizeof(float)>>>(dx.data().get(),dy.data().get(),blocks);
	hipDeviceSynchronize();
	double t2 = tim.lap_ms()/nreps;

	double gpu_sum = dx[0];  // D2H copy (1 word)
	printf("sum of %d numbers: host %.1f %.3f ms GPU %.1f %.3f ms\n",N,host_sum,t1,gpu_sum,t2);
	return 0;
}
